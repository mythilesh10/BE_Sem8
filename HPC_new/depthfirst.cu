
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;
const int MAXN = 1e5;
vector<int> adj[MAXN+5]; // adjacency list
bool visited[MAXN+5]; // mark visited nodes

void dfs(int node) {
    visited[node] = true;
    cout << node << " "; // Print the visited node here
    for (int i = 0; i < adj[node].size(); i++) {
        int next_node = adj[node][i];
        if (!visited[next_node]) {
            dfs(next_node);
        }
    }
}

int main() {
    cout << "Please enter nodes and edges: ";
    int n, m; // number of nodes and edges
    cin >> n >> m;
    for (int i = 1; i <= m; i++) {
        int u, v; // edge between u and v
        cin >> u >> v;
        adj[u].push_back(v);
        adj[v].push_back(u);
    }
    int start_node; // start node of DFS
    cout << "Enter the start node for DFS: ";
    cin >> start_node;
    dfs(start_node);
    cout << endl; // Print a newline after DFS traversal
    return 0;
}
